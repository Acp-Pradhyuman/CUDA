#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 10000000  // Define the size of the vectors
#define THREADS_PER_BLOCK 256

// CUDA kernel for partial dot product
__global__ void vector_dot_kernel(double *A, double *B, double *C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] * B[idx];  // Calculate individual products
    }
}

// CUDA kernel to sum the partial results
__global__ void reduce_sum(double *C, double *sum) {
    __shared__ double shared_data[THREADS_PER_BLOCK];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    if (idx < N) {
        shared_data[tid] = C[idx];
    } else {
        shared_data[tid] = 0.0;
    }
    __syncthreads();

    // Perform reduction within the block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    // Store the block's result in the global memory
    if (tid == 0) {
        atomicAdd(sum, shared_data[0]);
    }
}

int main() {
    double *A, *B, *C;
    double *d_A, *d_B, *d_C, *d_sum;
    double sum = 0.0;

    // Allocate memory for the vectors
    A = (double*)malloc(N * sizeof(double));
    B = (double*)malloc(N * sizeof(double));
    C = (double*)malloc(N * sizeof(double));

    // Initialize the vectors with random values
    for (int i = 0; i < N; i++) {
        // A[i] = rand() % 1000 + 1;  // Random values between 1 and 1000
        // B[i] = rand() % 1000 + 1;
        A[i] = i + 1;
        B[i] = N - i;
    }

    // Allocate device memory
    hipMalloc(&d_A, N * sizeof(double));
    hipMalloc(&d_B, N * sizeof(double));
    hipMalloc(&d_C, N * sizeof(double));
    hipMalloc(&d_sum, sizeof(double));

    // Copy input vectors from host to device
    hipMemcpy(d_A, A, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(double), hipMemcpyHostToDevice);

    // Initialize sum on device
    hipMemcpy(d_sum, &sum, sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel for partial dot product
    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    vector_dot_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_A, d_B, d_C);

    // Launch kernel to reduce the partial results
    reduce_sum<<<blocks, THREADS_PER_BLOCK>>>(d_C, d_sum);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float parallel_time = 0.0f;
    hipEventElapsedTime(&parallel_time, start, stop);

    // Copy the final sum from device to host
    hipMemcpy(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);

    printf("CUDA Dot Product: %f\n", sum);
    printf("CUDA Time: %f seconds\n", parallel_time / 1000.0);

    // Cleanup
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_sum);

    return 0;
}